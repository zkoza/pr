#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// zwykła funkcja w C/C++ 
void incrementArrayOnHost(double *tab, int N)
{
  for (int i=0; i < N; i++) 
    tab[i] += 1.0;
}

// funkcja (tzw. kernel) działająca na GPU
__global__ void incrementArrayOnDevice(double *tab, int N)
{
  int idx = blockIdx.y  * gridDim.x * blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
//  if (idx < N  && blockIdx.y > 0)
//    printf (" %d %d %d %d %d %d \n", idx,  gridDim.x, blockDim.x, N, blockIdx.x, threadIdx.x);
  if (idx<N) 
     tab[idx] += 1.0;
}



int main(void)
{
  const int N = 100000000;
  printf("N = %d\n", N);
  
  double *a_h, *b_h;     // wskaźniki na pamięć na CPU (host)
  double *a_d;           // wskaźnik na bufor w GPU (device)

  // przydział pamięci na CPU
  a_h = new double [N];
  b_h = new double [N];
  
  // przydział pamięci na GPU
  hipMalloc((void **) &a_d, sizeof(double)*N);
  
  // inicjalizacja danych na CPU
  for (int i=0; i<N; i++) 
  {
    a_h[i] = i + 1.0;
    b_h[i] = 0;
  }
  
  // przesłąnie danych na GPU: a_h ->  a_d
  hipMemcpy(a_d, a_h, sizeof(double)*N, hipMemcpyDefault );
  
  // robimy jakieś obliczenia na CPU
  incrementArrayOnHost(a_h, N);
  
  // a teraz próbujemy zrobić to samo na GPU
  dim3 blockSize = 512;
  dim3 gridSize (1,1,1);
  const int max_block_size = 65535;
  int nBlocks = N/blockSize.x + (N%blockSize.x == 0 ? 0 : 1);
  gridSize.y = 1 + nBlocks/max_block_size;
  gridSize.x = (nBlocks > max_block_size) ? max_block_size : nBlocks;

  printf("%d %d\n", gridSize.x, gridSize.y);

  // wywołujemy kernel na GPU 
  incrementArrayOnDevice <<< gridSize, blockSize >>> (a_d, N);
  // kopiujemy wynik z GPU do CPU
  hipDeviceSynchronize();
  hipMemcpy(b_h, a_d, sizeof(double)*N, hipMemcpyDefault);
  // sprawdzamy wynik
  for (int i=0; i<N; i++) 
  if (a_h[i] != b_h[i] && i < 100)
  {
    printf ("i=%d %g %g\n", i, a_h[i], b_h[i]);
    assert(a_h[i] == b_h[i]);
  }
 
  // sprzątamy
  delete [] a_h; 
  delete [] b_h;
  hipFree(a_d); 
  printf("Jeżeli widzisz ten napis, to program działa poprawnie\n");
}
