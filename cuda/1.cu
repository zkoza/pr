#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
int main(void)
{
  const int N = 1000;
  double *a_h, *b_h;     // pointers to host memory
  double *a_d, *b_d;     // pointers to device memory

  // allocate arrays on host
  a_h = new double [N];
  b_h = new double [N];
  
  // allocate arrays on device
  hipMalloc((void **) &a_d, sizeof(*a_d)*N);
  hipMalloc((void **) &b_d, sizeof(*a_d)*N);
  
  // initialize host data
  for (int i=0; i<N; i++) {
    a_h[i] = 10.0+i;
    b_h[i] = 0.0;
  }
  
  // send data from host to device: a_h to a_d
  hipMemcpy(a_d, a_h, sizeof(double)*N, hipMemcpyDefault );
  // copy data within device: a_d to b_d
  hipMemcpy(b_d, a_d, sizeof(double)*N, hipMemcpyDefault);
  // retrieve data from device: b_d to b_h
  hipMemcpy(b_h, b_d, sizeof(double)*N, hipMemcpyDefault);
  // check result
  for (int i=0; i<N; i++)
    assert(a_h[i] == b_h[i]);
  // cleanup
  delete [] a_h; 
  delete [] b_h;
  hipFree(a_d); 
  hipFree(b_d);
  printf("Jezeli widzisz ten napis, to program dziala poprawnie\n");
}
