#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// zwykła funkcja w C/C++ 
void incrementArrayOnHost(double *tab, int N)
{
  for (int i=0; i < N; i++) 
    tab[i] += 1.0;
}

// funkcja (tzw. kernel) działająca na GPU
__global__ void incrementArrayOnDevice(double *tab, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N) 
     tab[idx] += 1.0;
}



int main(void)
{
  const int N = 10000000;
  double *a_h, *b_h;     // wskaźniki na pamięć na CPU (host)
  double *a_d;           // wskaźnik na bufor w GPU (device)

  // przydział pamięci na CPU
  a_h = new double [N];
  b_h = new double [N];
  
  // przydział pamięci na GPU
  hipMalloc((void **) &a_d, sizeof(double)*N);
  
  // inicjalizacja danych na CPU
  for (int i=0; i<N; i++) 
  {
    a_h[i] = 1.0/(i + 1.0);
  }
  
  // przesłąnie danych na GPU: a_h ->  a_d
  hipMemcpy(a_d, a_h, sizeof(double)*N, hipMemcpyDefault );
  
  // robimy jakieś obliczenia na CPU
  incrementArrayOnHost(a_h, N);
  
  // a teraz próbujemy zrobić to samo na GPU
  int blockSize = 320;
  int nBlocks = N/blockSize + (N%blockSize == 0 ? 0 : 1);
  // wywołujemy kernel na GPU 
  incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N);
  // kopiujemy wynik z GPU do CPU
  hipMemcpy(b_h, a_d, sizeof(double)*N, hipMemcpyDefault);
  // sprawdzamy wynik
  for (int i=0; i<N; i++) 
     assert(a_h[i] == b_h[i]);
 
  // sprzątamy
  delete [] a_h; 
  delete [] b_h;
  hipFree(a_d); 
  printf("Jeżeli widzisz ten napis, to program działa poprawnie\n");
}
