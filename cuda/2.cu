#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define CHECK_CUDA(x)           \
{                               \
  hipError_t err = x;          \
  if (err != hipSuccess)       \
  {                             \
     printf("!!! CUDA ERROR: \"%s\" at file %s, line %d !!!\n", hipGetErrorString(err), __FILE__, __LINE__);\
     exit(1);                   \
  }                             \
}

int main(int argc, char* argv[])
{
  const int N = (argc==1)?  10000 : atoi(argv[1]);
  printf("N = %d\n", N);

  double *a_h, *b_h;     // pointers to host memory
  double *a_d, *b_d;     // pointers to device memory

  // allocate arrays on host
  a_h = new double [N];
  b_h = new double [N];
  
  // allocate arrays on device
  CHECK_CUDA( hipMalloc((void **) &a_d, sizeof(double)*N) );
  CHECK_CUDA( hipMalloc((void **) &b_d, sizeof(double)*N) );
  
  // initialize host data
  for (int i=0; i<N; i++) {
    a_h[i] = 10.f+i;
    b_h[i] = 0.f;
  }
  
  // send data from host to device: a_h to a_d
  CHECK_CUDA( hipMemcpy(a_d, a_h, sizeof(double)*N, hipMemcpyDefault) );
  // copy data within device: a_d to b_d
  CHECK_CUDA( hipMemcpy(b_d, a_d, sizeof(double)*N, hipMemcpyDefault) );
  // retrieve data from device: b_d to b_h
  CHECK_CUDA( hipMemcpy(b_h, b_d, sizeof(double)*N, hipMemcpyDefault) );
  // check result
  for (int i=0; i<N; i++)
    assert(a_h[i] == b_h[i]);
  // cleanup
  delete [] a_h; 
  delete [] b_h;
  CHECK_CUDA( hipFree(a_d) ); 
  CHECK_CUDA( hipFree(b_d) );
  printf("Gratulacje, program dziala poprawnie!!!\n");
}
